#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

#define TILE_DIM 32
#define BLOCK_ROWS 8

typedef float dtype;

__global__ 
void matTrans(dtype* AT, dtype* A, int N)  {
	/* Fill your code here */
	__shared__ dtype tile[TILE_DIM][TILE_DIM+1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y+j][threadIdx.x] = A[(y+j)*N + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;  
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        AT[(y+j)*N + x] = tile[threadIdx.x][threadIdx.y + j];

}

void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}

void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}

void
gpuTranspose (dtype* A, dtype* AT, int N)
{
  struct stopwatch_t* timer = NULL;
  long double t_gpu;
  stopwatch_init ();
  timer = stopwatch_create ();
  
  stopwatch_start (timer);

	/* run your kernel here */
	dtype *d_A, *d_AT;
	CUDA_CHECK_ERROR(hipMalloc((void**)&d_A, N * N * sizeof(dtype)));
	CUDA_CHECK_ERROR(hipMalloc((void**)&d_AT, N * N * sizeof(dtype)));

	CUDA_CHECK_ERROR(hipMemcpy(d_A, A, N * N * sizeof(dtype), hipMemcpyHostToDevice));

	dim3 blockDims(TILE_DIM, BLOCK_ROWS, 1);
	dim3 gridDims((N + TILE_DIM - 1) / TILE_DIM, (N + TILE_DIM - 1) / TILE_DIM, 1);

	matTrans<<<gridDims, blockDims>>>(d_AT, d_A, N);

	CUDA_CHECK_ERROR(hipMemcpy(AT, d_AT, N * N * sizeof(dtype), hipMemcpyDeviceToHost));

	CUDA_CHECK_ERROR(hipFree(d_A));
	CUDA_CHECK_ERROR(hipFree(d_AT));

	/*end of my code*/

  hipDeviceSynchronize ();
  t_gpu = stopwatch_stop (timer);
  fprintf (stdout, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
           t_gpu, (N * N) / t_gpu * 1e-9 );

}

int 
main(int argc, char** argv)
{
	dtype *A, *ATgpu, *ATcpu;
  int err;

	int N;

  struct stopwatch_t* timer = NULL;
  long double t_cpu;

	N = -1;
	parseArg (argc, argv, &N);

  ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
  ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

  A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	gpuTranspose (A, ATgpu, N);

  stopwatch_init ();
  timer = stopwatch_create ();

	stopwatch_start (timer);
	cpuTranspose (A, ATcpu, N);
  t_cpu = stopwatch_stop (timer);
  fprintf (stdout, "Time to execute CPU transpose kernel: %Lg secs\n",
           t_cpu);

	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stdout, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}
